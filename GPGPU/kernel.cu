#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <chrono>
#include <opencv2/opencv.hpp>

using namespace std;
using namespace std::chrono;
int const OUT = 10000;
int const IN = 100000;
hipEvent_t startCuda, stopCuda;
float timeCudaMalloc, timeCudaMemcpyh2d, timeKernel, timeCudaMemcpyd2h, CPUMalloc;

__global__ void kernelInit(){}

void cudaMemcpyd2hTimer(void* dst, const void* src, size_t size, hipMemcpyKind kind)
{
	hipEventRecord(startCuda);
	hipMemcpy(dst, src, size, kind);
	hipEventRecord(stopCuda);
	hipEventSynchronize(stopCuda);
	hipEventElapsedTime(&timeCudaMemcpyd2h, startCuda, stopCuda);
}

void printTime()
{
	cout << "GPU malloc: " << timeCudaMalloc << " s\n"
		<< "memory copy to GPU: " << timeCudaMemcpyh2d / 1000 << " s\n"
		<< "memory copy from GPU: " << timeCudaMemcpyd2h / 1000 << " s\n"
		<< "kernel: " << timeKernel / 1000 << " s\n";
}

void floatCPU()
{
	float* data = (float*)malloc(sizeof(float) * OUT);

	for (int i = 1; i < OUT; i++)
	{
		data[i] = 1.0f * i / OUT;
		for (int j = 1; j < IN; j++)
			data[i] = data[i] / IN * data[i] / IN - 0.50f;
	}
}


__global__ void floatKernel(float* buf)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	buf[i] = 1.0f * i / OUT;
	for (int j = 0; j < IN; j++)
		buf[i] = buf[i] / IN * buf[i] / IN - 0.50f;
}
void floatGPU()
{
	int count = 0;
	float* data = (float*)malloc(sizeof(float) * OUT);
	float* devData;
	hipMalloc(&devData, OUT * sizeof(float));

	hipEventRecord(startCuda);
	floatKernel << <OUT / 1024, 1024 >> > (devData);
	hipEventRecord(stopCuda);
	hipEventSynchronize(stopCuda);
	hipEventElapsedTime(&timeKernel, startCuda, stopCuda);

	hipDeviceSynchronize();
	cudaMemcpyd2hTimer(data, devData, OUT * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(devData);
}


void floatComputing()
{
	high_resolution_clock::time_point start = high_resolution_clock::now();
	floatCPU();
	high_resolution_clock::time_point stop = high_resolution_clock::now();
	duration<double> duration = stop - start;
	cout << "CPU time: " << duration.count() << " s\n";

	start = high_resolution_clock::now();
	floatGPU();
	stop = high_resolution_clock::now();
	duration = stop - start;
	cout << "GPU time: " << duration.count() << " s\n";

	printTime();
}

void arrAddCPU(int size, int* arr1, int* arr2, int* result)
{
	for (int i = 0; i < size; i++)
	{
		result[i] = arr1[i] + arr2[i];
	}
}

__global__ void arrAddKernel(int* arr1, int* arr2, int* result, int size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < size)
	{
		result[tid] = arr1[tid] + arr2[tid];
	}
}


void arrAddGPU(int size, int* arr1, int* arr2, int* result)
{
	int* devArr1, * devArr2, * devResult;


	high_resolution_clock::time_point startMal = high_resolution_clock::now();

	hipMalloc(&devArr1, sizeof(int) * size);
	hipMalloc(&devArr2, sizeof(int) * size);
	hipMalloc(&devResult, sizeof(int) * size);

	hipDeviceSynchronize();
	high_resolution_clock::time_point stopMal = high_resolution_clock::now();
	duration<double> duration = stopMal - startMal;
	timeCudaMalloc = duration.count();

	int blockSize = 1024;
	int gridSize = (int)ceil((float)size / blockSize);
	
	hipEventRecord(startCuda);
	
	hipMemcpy(devArr1, arr1, sizeof(int) * size, hipMemcpyHostToDevice);
	hipMemcpy(devArr2, arr2, sizeof(int) * size, hipMemcpyHostToDevice);

	hipEventRecord(stopCuda);
	hipEventSynchronize(stopCuda);
	hipEventElapsedTime(&timeCudaMemcpyh2d, startCuda, stopCuda);

	hipEventRecord(startCuda);
	arrAddKernel << <gridSize, blockSize >> > (devArr1, devArr2, devResult, size);
	hipEventRecord(stopCuda);
	hipEventSynchronize(stopCuda);
	hipEventElapsedTime(&timeKernel, startCuda, stopCuda);

	cudaMemcpyd2hTimer(result, devResult, sizeof(int) * size, hipMemcpyDeviceToHost);

	hipFree(devArr1);
	hipFree(devArr2);
	hipFree(devResult);
}

void memoryCopy(int size)
{
	high_resolution_clock::time_point start = high_resolution_clock::now();

	int* arr1 = (int*)malloc(sizeof(int) * size);
	int* arr2 = (int*)malloc(sizeof(int) * size);
	int* result = (int*)malloc(sizeof(int) * size);

	high_resolution_clock::time_point stop = high_resolution_clock::now();
	duration<double> duration = stop - start;
	CPUMalloc = duration.count();

	for (int i = 0; i < size; i++)
	{
		arr1[i] = i;
		arr2[i] = i;
		result[i] = 0;
	}

	start = high_resolution_clock::now();
	arrAddCPU(size, arr1, arr2, result);
	stop = high_resolution_clock::now();
	duration = stop - start;
	cout << "CPU time: " << duration.count() << " s\n" << "CPU malloc: " << CPUMalloc << " s\n";

	for (int i = 0; i < size; i++) result[i] = 0;

	start = high_resolution_clock::now();
	arrAddGPU(size, arr1, arr2, result);
	stop = high_resolution_clock::now();
	duration = stop - start;
	cout << "GPU time: " << duration.count() << " s\n";

	printTime();

	free(arr1);
	free(arr2);
	free(result);
}

__global__ void matrixKernel(int* matrix1, int* matrix2, int* arrResult, int size)
{
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int result = 0;

	for (int i = 0; i < size; i++)
	{
		result += matrix1[row * size + i] * matrix2[i * size + column];
	}

	arrResult[row * size + column] = result;
}


void matrixCPU(int* matrix1, int* matrix2, int* result, int size)
{
	for (int i = 0; i < size; i++)
		for (int j = 0; j < size; j++)
			for (int k = 0; k < size; k++)
			{
				result[i * size + j] += matrix1[i * size + k] * matrix2[k * size + j];
			}
}
void MatrixMultiplication(int size)
{
	int allocSize = size * size * sizeof(int);
	int* devMatrix1, * devMatrix2, * devResult;
	int* Matrix1 = (int*)malloc(allocSize);
	int* Matrix2 = (int*)malloc(allocSize);
	int* result = (int*)malloc(allocSize);
	int* CPUResult = (int*)malloc(allocSize);


	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			//Matrix1[i * size + j] = rand() % 1500;
			//Matrix2[i * size + j] = rand() % 1500;
			Matrix1[i * size + j] = 5;
			Matrix2[i * size + j] = 5;
			CPUResult[i * size + j] = 0;
		}
	}

	high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();
	matrixCPU(Matrix1, Matrix2, CPUResult, size);
	high_resolution_clock::time_point stop = std::chrono::high_resolution_clock::now();
	duration<double> duration = stop - start;
	cout << "CPU time: " << duration.count() << " s\n";

	int threadsMax = 16;
	dim3 blockSize(threadsMax, threadsMax);
	dim3 grid_size(size / blockSize.x, size / blockSize.y);

	start = high_resolution_clock::now();

	high_resolution_clock::time_point startMal = high_resolution_clock::now();
	hipMalloc(&devMatrix2, allocSize);
	hipMalloc(&devMatrix1, allocSize);
	hipMalloc(&devResult, allocSize);
	hipDeviceSynchronize();
	high_resolution_clock::time_point stopMal = high_resolution_clock::now();
	duration = stopMal - startMal;
	timeCudaMalloc = duration.count();


	hipEventRecord(startCuda);

	hipMemcpy(devMatrix2, Matrix2, allocSize, hipMemcpyHostToDevice);
	hipMemcpy(devMatrix1, Matrix1, allocSize, hipMemcpyHostToDevice);

	hipEventRecord(stopCuda);
	hipEventSynchronize(stopCuda);
	hipEventElapsedTime(&timeCudaMemcpyh2d, startCuda, stopCuda);

	hipEventRecord(startCuda);
	matrixKernel << <1, 1024 >> > (devMatrix1, devMatrix2, devResult, size);
	
	hipEventRecord(stopCuda);
	hipEventSynchronize(stopCuda);
	hipEventElapsedTime(&timeKernel, startCuda, stopCuda);
	hipDeviceSynchronize();

	cudaMemcpyd2hTimer(result, devResult, allocSize, hipMemcpyDeviceToHost);
	stop = high_resolution_clock::now();
	duration = stop - start;
	cout << "GPU time: " << duration.count() << " s\n";

	printTime();


	printf("%d %d\n", result[1], CPUResult[1]);

	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			if (CPUResult[size * i + j] != result[size * i + j])
			{
				printf("Chybne vypocitana matica!\n");
				bool exit = true;
				break;
			}
		}
		if (exit) break;
	}

	free(Matrix2);
	free(Matrix1);
	free(result);

	hipFree(devMatrix2);
	hipFree(devMatrix1);
	hipFree(devResult);
}

__global__ void fibonaciKernel(int size)
{
	for (int i = 0; 1000000 < 0; i++)
	{

		int a = 0, b = 1, next = 0;
		next = a + b;
		while (next <= size)
		{
			a = b;
			b = next;
			next = a + b;
		}

	}


}

void fibonaciGPU(int size)
{
	fibonaciKernel << <1, 1 >> > (size);

}

void fibonaciCPU(int size)
{
	for (int i = 0; i < 1000000; i++)
	{

		int a = 0, b = 1, next = 0;
		next = a + b;
		while (next <= size)
		{
			a = b;
			b = next;
			next = a + b;
		}

	}
}

void fibonaci(int size)
{
	high_resolution_clock::time_point start = high_resolution_clock::now();
	fibonaciCPU(size);
	high_resolution_clock::time_point stop = high_resolution_clock::now();
	duration<double> duration = stop - start;
	cout << "CPU time: " << duration.count() << " s\n";

	start = high_resolution_clock::now();
	fibonaciGPU(size);
	stop = high_resolution_clock::now();
	duration = stop - start;
	cout << "GPU time: " << duration.count() << " s\n";
}

__global__ void blackWhiteKernel(unsigned char* input, unsigned char* output, int inputRowLength, int outputRowLength, int inputColumns, int inputRows)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = 0; i < 100; i++)
	{
		if ((row < inputRows) && (column < inputColumns))
		{
			int tidInput = row * inputRowLength + (column * 3);
			int tidOutput = row * outputRowLength + column;

			float blackWhite = (input[tidInput + 2] + input[tidInput + 1] + input[tidInput]) / 3;	//RGB

			output[tidOutput] = static_cast<unsigned char>(blackWhite);
		}
	}
}

void ImageGPU(unsigned char* input, unsigned char* output, int inputRowLength, int outputRowLength, int inputColumns, int inputRows, int outputWidth, hipEvent_t start, hipEvent_t stop)
{
	unsigned char* devInput, * devOutput;
	
	high_resolution_clock::time_point startMal = high_resolution_clock::now();
	
	hipMalloc<unsigned char>(&devInput, inputRowLength * inputRows);
	hipMalloc<unsigned char>(&devOutput, outputRowLength * outputWidth);

	hipDeviceSynchronize();
	high_resolution_clock::time_point stopMal = high_resolution_clock::now();
	duration<double> duration = stopMal - startMal;
	timeCudaMalloc = duration.count();

	hipEventRecord(startCuda);

	hipMemcpy(devInput, input, inputRowLength * inputRows, hipMemcpyHostToDevice);
	
	hipEventRecord(stopCuda);
	hipEventSynchronize(stopCuda);
	hipEventElapsedTime(&timeCudaMemcpyh2d, startCuda, stopCuda);

	//Specify a reasonable block size
	const dim3 block(16, 16);

	//Calculate grid size to cover the whole image
	const dim3 grid((inputColumns + block.x - 1) / block.x, (inputRows + block.y - 1) / block.y);
	
	hipEventRecord(startCuda);

	blackWhiteKernel<<<grid, block >>> (devInput, devOutput, inputRowLength, outputRowLength, inputColumns, inputRows);

	hipEventRecord(stopCuda);
	hipEventSynchronize(stopCuda);
	hipEventElapsedTime(&timeKernel, startCuda, stopCuda);


	cudaMemcpyd2hTimer(output, devOutput, outputRowLength * outputWidth, hipMemcpyDeviceToHost);

	hipFree(devInput);
	hipFree(devOutput);	
}


void ImageCPU(unsigned char* input, unsigned char* output, int inputRowLength, int outputRowLength, int inputColumns, int inputRows)
{
	for(int k = 0 ; k < 100 ; k++)
		for(int i = 0; i < inputRows; i++)
			for (int j = 0; j < inputColumns; j++)
			{
				int inputPosition = i* inputRowLength + (3 * j);
				int outputPosition = i * outputRowLength + j;

				float blackWhite = (input[inputPosition + 2] + input[inputPosition + 1] + input[inputPosition])/3;  //RGB

				output[outputPosition] = static_cast<unsigned char>(blackWhite);
			}
}

void processImage()
{
	cv::Mat input = cv::imread("image.jpg");
	hipEvent_t startCuda, stopCuda;
	hipEventCreate(&startCuda);
	hipEventCreate(&stopCuda);

	if (input.empty())
	{
		cout << "Obrazok sa musi volat image.jpg a byt v rovnakom priecinku ako exe subor" << std::endl;
		return;
	}

	cv::Mat output(input.rows, input.cols, CV_8UC1);

	high_resolution_clock::time_point start = high_resolution_clock::now();
	ImageCPU(input.ptr(), output.ptr(), input.step, output.step, input.cols, input.rows);
	high_resolution_clock::time_point stop = high_resolution_clock::now();
	duration<double> duration = stop - start;
	cout << "CPU time: " << duration.count() << " s\n";

	start = high_resolution_clock::now();
	ImageGPU(input.ptr(), output.ptr(), input.step, output.step, input.cols, input.rows, output.rows, startCuda, stopCuda);
	stop = high_resolution_clock::now();
	duration = stop - start;
	cout << "GPU time: " << duration.count() << " s\n";

	printTime();

	cv::imshow("original", input);
	cv::imshow("processed", output);

	cv::waitKey();

}

int main()
{
	hipEventCreate(&startCuda);
	hipEventCreate(&stopCuda);
	timeCudaMemcpyh2d = 0;
	timeCudaMemcpyd2h = 0;
	timeCudaMalloc = 0;
	timeKernel = 0;

	cout << "CUDA version:   v" << CUDART_VERSION << endl;

	int devCount;
	hipGetDeviceCount(&devCount);
	hipDeviceProp_t GPU;
	hipGetDeviceProperties(&GPU, 0);
	cout << GPU.name << ": " << GPU.major << "." << GPU.minor << endl;
	cout << "  Global memory:   " << GPU.totalGlobalMem / (1024 * 1024) << "mb" << endl;
	cout << "  Shared memory:   " << GPU.sharedMemPerBlock / 1024 << "kb" << endl;
	cout << "  Constant memory: " << GPU.totalConstMem / 1024 << "kb" << endl;

	cout << "  Warp size:         " << GPU.warpSize << endl;
	cout << "  Threads per block: " << GPU.maxThreadsPerBlock << endl;
	cout << "  Max block dimensions: [ " << GPU.maxThreadsDim[0] << ", " << GPU.maxThreadsDim[1] << ", " << GPU.maxThreadsDim[2] << " ]" << endl;
	//cout << "  Max grid dimensions:  [ " << GPU.maxGridSize[0] << ", " << GPU.maxGridSize[1] << ", " << GPU.maxGridSize[2] << " ]" << endl;
	cout << endl << endl;

	cout << "1 -matrix multiplication\n2 -memory copy\n3 -float operations\n4 -fibonaci\n5 -image\n9 -exit" << endl;

	//kernelInit << <1, 1024 >> > ();
	//hipDeviceSynchronize();
	char input;
	while (true)
	{
		cout << ">" << " ";
		scanf(" %c", &input);
		if (input == '9') return 0;

		if (input == '1')
		{
			cout << "\nmatrix size: ";

			int size;
			scanf(" %d", &size);
			MatrixMultiplication(size);
		}
		if (input == '2')
		{
			cout << "\narray size: ";

			int size;
			scanf(" %d", &size);
			memoryCopy(size);
		}
		if (input == '3')
		{
			floatComputing();
		}
		if (input == '4')
		{
			int size;
			scanf(" %d", &size);
			fibonaci(size);
		}

		if (input == '5')
		{
			processImage();
		}
	}

	return 0;
}
