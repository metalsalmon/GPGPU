#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <chrono>
#include <opencv2/opencv.hpp>
#include <sstream>
#include <fstream>

using namespace std;
using namespace std::chrono;
int const OUT = 10000;
int IN = 10000;
hipEvent_t startCuda, stopCuda;
float timeCudaMalloc, timeCudaMemcpyh2d, timeKernel, timeCudaMemcpyd2h, CPUMalloc;
stringstream ss;

struct Settings {
	int	test1Base, test1Increment;
	int	test2Base, test2Increment;
	int	test3Base, test3Increment;
	int	test4Base, test4Increment;
	int	test5Base, test5Increment;
	int repeat;
};

__global__ void kernelInit(){}

void readConfig(Settings& settings)
{
	ifstream fileConfig("config.txt");
	string line;
	while (getline(fileConfig, line)) {
		istringstream readline(line.substr(line.find(":") + 1));
		if (line.find("repeat") != -1) readline >> settings.repeat;
		else if (line.find("base1") != -1) readline >> settings.test1Base;
		else if (line.find("base2") != -1) readline >> settings.test2Base;
		else if (line.find("base3") != -1) readline >> settings.test3Base;
		else if (line.find("base4") != -1) readline >> settings.test4Base;
		else if (line.find("base5") != -1) readline >> settings.test5Base;
		else if (line.find("increment1") != -1) readline >> settings.test1Increment;
		else if (line.find("increment2") != -1) readline >> settings.test2Increment;
		else if (line.find("increment3") != -1) readline >> settings.test3Increment;
		else if (line.find("increment4") != -1) readline >> settings.test4Increment;
		else if (line.find("increment5") != -1) readline >> settings.test5Increment;
	}
	fileConfig.close();
}

void cudaMemcpyd2hTimer(void* dst, const void* src, size_t size, hipMemcpyKind kind)
{
	hipEventRecord(startCuda);
	hipMemcpy(dst, src, size, kind);
	hipEventRecord(stopCuda);
	hipEventSynchronize(stopCuda);
	hipEventElapsedTime(&timeCudaMemcpyd2h, startCuda, stopCuda);
}

void printTime()
{
	cout <<"GPU malloc: " << timeCudaMalloc << " s\n"
		<< "memory copy to GPU: " << timeCudaMemcpyh2d / 1000 << " s\n"
		<< "memory copy from GPU: " << timeCudaMemcpyd2h / 1000 << " s\n"
		<< "kernel execution time: " << timeKernel / 1000 << " s\n\n";

	ss  << "GPU malloc: " << timeCudaMalloc << " s\n"
		<< "memory copy to GPU: " << timeCudaMemcpyh2d / 1000 << " s\n"
		<< "memory copy from GPU: " << timeCudaMemcpyd2h / 1000 << " s\n"
		<< "kernel execution time: " << timeKernel / 1000 << " s\n\n";


}

void CPUGPUTime(string text)
{
	cout << text;
	ss << text;
}

void printTestNumber(string text)
{
	cout << text;
	ss << text;
}

void floatCPU()
{
	float* data = (float*)malloc(sizeof(float) * OUT);

	for (int i = 1; i < OUT; i++)
	{
		data[i] = 1.0f * i / OUT;
		for (int j = 1; j < IN; j++)
			data[i] = data[i] / IN * data[i] / IN - 0.50f;
	}
}


__global__ void floatKernel(float* buf, int IN)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	buf[i] = 1.0f * i / OUT;
	for (int j = 0; j < IN; j++)
		buf[i] = buf[i] / IN * buf[i] / IN - 0.50f;
}
void floatGPU()
{
	float* data = (float*)malloc(sizeof(float) * OUT);
	float* devData;

	high_resolution_clock::time_point startMal = high_resolution_clock::now();

	hipMalloc(&devData, OUT * sizeof(float));


	hipDeviceSynchronize();
	high_resolution_clock::time_point stopMal = high_resolution_clock::now();
	duration<double> duration = stopMal - startMal;
	timeCudaMalloc = duration.count();

	hipEventRecord(startCuda);
	floatKernel << <OUT / 1024, 1024 >> > (devData, IN);
	hipEventRecord(stopCuda);
	hipEventSynchronize(stopCuda);
	hipEventElapsedTime(&timeKernel, startCuda, stopCuda);

	hipDeviceSynchronize();
	cudaMemcpyd2hTimer(data, devData, OUT * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(devData);
}


void floatComputing()
{
	high_resolution_clock::time_point start = high_resolution_clock::now();
	floatCPU();
	high_resolution_clock::time_point stop = high_resolution_clock::now();
	duration<double> duration = stop - start;
	CPUGPUTime("CPU time: " + to_string(duration.count()) + " s\n");

	start = high_resolution_clock::now();
	floatGPU();
	stop = high_resolution_clock::now();
	duration = stop - start;
	CPUGPUTime("GPU time: " + to_string(duration.count()) + " s\n");

	printTime();
}

void arrAddCPU(int size, int* arr1, int* arr2, int* result)
{
	for (int i = 0; i < size; i++)
	{
		result[i] = arr1[i] + arr2[i];
	}
}

__global__ void arrAddKernel(int* arr1, int* arr2, int* result, int size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < size)
	{
		result[tid] = arr1[tid] + arr2[tid];
	}
}


void arrAddGPU(int size, int* arr1, int* arr2, int* result)
{
	int* devArr1, * devArr2, * devResult;


	high_resolution_clock::time_point startMal = high_resolution_clock::now();

	hipMalloc(&devArr1, sizeof(int) * size);
	hipMalloc(&devArr2, sizeof(int) * size);
	hipMalloc(&devResult, sizeof(int) * size);

	hipDeviceSynchronize();
	high_resolution_clock::time_point stopMal = high_resolution_clock::now();
	duration<double> duration = stopMal - startMal;
	timeCudaMalloc = duration.count();

	int blockSize = 1024;
	int gridSize = (int)ceil((float)size / blockSize);
	
	hipEventRecord(startCuda);
	
	hipMemcpy(devArr1, arr1, sizeof(int) * size, hipMemcpyHostToDevice);
	hipMemcpy(devArr2, arr2, sizeof(int) * size, hipMemcpyHostToDevice);

	hipEventRecord(stopCuda);
	hipEventSynchronize(stopCuda);
	hipEventElapsedTime(&timeCudaMemcpyh2d, startCuda, stopCuda);

	hipEventRecord(startCuda);
	arrAddKernel << <gridSize, blockSize >> > (devArr1, devArr2, devResult, size);
	hipEventRecord(stopCuda);
	hipEventSynchronize(stopCuda);
	hipEventElapsedTime(&timeKernel, startCuda, stopCuda);

	cudaMemcpyd2hTimer(result, devResult, sizeof(int) * size, hipMemcpyDeviceToHost);

	hipFree(devArr1);
	hipFree(devArr2);
	hipFree(devResult);
}

void memoryCopy(int size)
{
	high_resolution_clock::time_point start = high_resolution_clock::now();

	int* arr1 = (int*)malloc(sizeof(int) * size);
	int* arr2 = (int*)malloc(sizeof(int) * size);
	int* result = (int*)malloc(sizeof(int) * size);

	high_resolution_clock::time_point stop = high_resolution_clock::now();
	duration<double> duration = stop - start;
	CPUMalloc = duration.count();

	for (int i = 0; i < size; i++)
	{
		arr1[i] = i;
		arr2[i] = i;
		result[i] = 0;
	}

	start = high_resolution_clock::now();
	arrAddCPU(size, arr1, arr2, result);
	stop = high_resolution_clock::now();
	duration = stop - start;
	CPUGPUTime("CPU time: " + to_string(duration.count()) + " s\n" + "CPU malloc: " +to_string(CPUMalloc) + " s\n");

	for (int i = 0; i < size; i++) result[i] = 0;

	start = high_resolution_clock::now();
	arrAddGPU(size, arr1, arr2, result);
	stop = high_resolution_clock::now();
	duration = stop - start;
	CPUGPUTime("GPU time: " + to_string(duration.count()) + " s\n");

	printTime();

	free(arr1);
	free(arr2);
	free(result);
}

__global__ void matrixKernel(int* matrix1, int* matrix2, int* arrResult, int size)
{
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int result = 0;

	for (int i = 0; i < size; i++)
	{
		result += matrix1[row * size + i] * matrix2[i * size + column];
	}

	arrResult[row * size + column] = result;
}


void matrixCPU(int* matrix1, int* matrix2, int* result, int size)
{
	for (int i = 0; i < size; i++)
		for (int j = 0; j < size; j++)
			for (int k = 0; k < size; k++)
			{
				result[i * size + j] += matrix1[i * size + k] * matrix2[k * size + j];
			}
}
void MatrixMultiplication(int size)
{
	int allocSize = size * size * sizeof(int);
	int* devMatrix1, * devMatrix2, * devResult;
	int* Matrix1 = (int*)malloc(allocSize);
	int* Matrix2 = (int*)malloc(allocSize);
	int* result = (int*)malloc(allocSize);
	int* CPUResult = (int*)malloc(allocSize);


	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			//Matrix1[i * size + j] = rand() % 1500;
			//Matrix2[i * size + j] = rand() % 1500;
			Matrix1[i * size + j] = 5;
			Matrix2[i * size + j] = 5;
			CPUResult[i * size + j] = 0;
		}
	}

	high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();
	matrixCPU(Matrix1, Matrix2, CPUResult, size);
	high_resolution_clock::time_point stop = std::chrono::high_resolution_clock::now();
	duration<double> duration = stop - start;
	CPUGPUTime("CPU time: " + to_string(duration.count()) + " s\n");

	int threadsMax = 16;
	dim3 blockSize(threadsMax, threadsMax);
	dim3 grid_size(size / blockSize.x, size / blockSize.y);

	start = high_resolution_clock::now();

	high_resolution_clock::time_point startMal = high_resolution_clock::now();
	hipMalloc(&devMatrix2, allocSize);
	hipMalloc(&devMatrix1, allocSize);
	hipMalloc(&devResult, allocSize);
	hipDeviceSynchronize();
	high_resolution_clock::time_point stopMal = high_resolution_clock::now();
	duration = stopMal - startMal;
	timeCudaMalloc = duration.count();


	hipEventRecord(startCuda);

	hipMemcpy(devMatrix2, Matrix2, allocSize, hipMemcpyHostToDevice);
	hipMemcpy(devMatrix1, Matrix1, allocSize, hipMemcpyHostToDevice);

	hipEventRecord(stopCuda);
	hipEventSynchronize(stopCuda);
	hipEventElapsedTime(&timeCudaMemcpyh2d, startCuda, stopCuda);

	hipEventRecord(startCuda);
	matrixKernel << <1, 1024 >> > (devMatrix1, devMatrix2, devResult, size);
	
	hipEventRecord(stopCuda);
	hipEventSynchronize(stopCuda);
	hipEventElapsedTime(&timeKernel, startCuda, stopCuda);
	hipDeviceSynchronize();

	cudaMemcpyd2hTimer(result, devResult, allocSize, hipMemcpyDeviceToHost);
	stop = high_resolution_clock::now();
	duration = stop - start;
	CPUGPUTime("GPU time: " + to_string(duration.count()) + " s\n");

	printTime();


	//printf("%d %d\n", result[1], CPUResult[1]);

	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			if (CPUResult[size * i + j] != result[size * i + j])
			{
				cout << "Chybne vypocitana matica!\n";
				bool exit = true;
				break;
			}
		}
		if (exit) break;
	}

	free(Matrix2);
	free(Matrix1);
	free(result);

	hipFree(devMatrix2);
	hipFree(devMatrix1);
	hipFree(devResult);
}

__global__ void fibonaciKernel(int size)
{
	for (int i = 0; 1000000 < 0; i++)
	{

		int a = 0, b = 1, next = 0;
		next = a + b;
		while (next <= size)
		{
			a = b;
			b = next;
			next = a + b;
		}

	}


}

void fibonaciGPU(int size)
{
	fibonaciKernel << <1, 1 >> > (size);

}

void fibonaciCPU(int size)
{
	for (int i = 0; i < 1000000; i++)
	{

		int a = 0, b = 1, next = 0;
		next = a + b;
		while (next <= size)
		{
			a = b;
			b = next;
			next = a + b;
		}

	}
}

void fibonaci(int size)
{
	high_resolution_clock::time_point start = high_resolution_clock::now();
	fibonaciCPU(size);
	high_resolution_clock::time_point stop = high_resolution_clock::now();
	duration<double> duration = stop - start;
	CPUGPUTime("CPU time: " + to_string(duration.count()) + " s\n");

	start = high_resolution_clock::now();
	fibonaciGPU(size);
	stop = high_resolution_clock::now();
	duration = stop - start;
	CPUGPUTime("GPU time: " + to_string(duration.count()) + " s\n");
}

__global__ void blackWhiteKernel(unsigned char* input, unsigned char* output, int inputRowLength, int outputRowLength, int inputColumns, int inputRows, int size)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = 0; i < size; i++)
	{
		if ((row < inputRows) && (column < inputColumns))
		{
			int tidInput = row * inputRowLength + (column * 3);
			int tidOutput = row * outputRowLength + column;

			float blackWhite = (input[tidInput + 2] + input[tidInput + 1] + input[tidInput]) / 3;	//RGB

			output[tidOutput] = static_cast<unsigned char>(blackWhite);
		}
	}
}

void ImageGPU(unsigned char* input, unsigned char* output, int inputRowLength, int outputRowLength, int inputColumns, int inputRows, int outputWidth, int size)
{
	unsigned char* devInput, * devOutput;
	
	high_resolution_clock::time_point startMal = high_resolution_clock::now();
	
	hipMalloc<unsigned char>(&devInput, inputRowLength * inputRows);
	hipMalloc<unsigned char>(&devOutput, outputRowLength * outputWidth);

	hipDeviceSynchronize();
	high_resolution_clock::time_point stopMal = high_resolution_clock::now();
	duration<double> duration = stopMal - startMal;
	timeCudaMalloc = duration.count();

	hipEventRecord(startCuda);

	hipMemcpy(devInput, input, inputRowLength * inputRows, hipMemcpyHostToDevice);
	
	hipEventRecord(stopCuda);
	hipEventSynchronize(stopCuda);
	hipEventElapsedTime(&timeCudaMemcpyh2d, startCuda, stopCuda);

	//Specify a reasonable block size
	const dim3 block(32, 32);

	//Calculate grid size to cover the whole image
	const dim3 grid((inputColumns + block.x - 1) / block.x, (inputRows + block.y - 1) / block.y);
	
	hipEventRecord(startCuda);

	blackWhiteKernel<<<grid, block >>> (devInput, devOutput, inputRowLength, outputRowLength, inputColumns, inputRows, size);

	hipEventRecord(stopCuda);
	hipEventSynchronize(stopCuda);
	hipEventElapsedTime(&timeKernel, startCuda, stopCuda);


	cudaMemcpyd2hTimer(output, devOutput, outputRowLength * outputWidth, hipMemcpyDeviceToHost);

	hipFree(devInput);
	hipFree(devOutput);	
}


void ImageCPU(unsigned char* input, unsigned char* output, int inputRowLength, int outputRowLength, int inputColumns, int inputRows, int size)
{
	for(int k = 0 ; k < size ; k++)
		for(int i = 0; i < inputRows; i++)
			for (int j = 0; j < inputColumns; j++)
			{
				int inputPosition = i* inputRowLength + (3 * j);
				int outputPosition = i * outputRowLength + j;

				float blackWhite = (input[inputPosition + 2] + input[inputPosition + 1] + input[inputPosition])/3;  //RGB

				output[outputPosition] = static_cast<unsigned char>(blackWhite);
			}
}

void processImage(int size)
{
	cv::Mat input = cv::imread("image.jpg");
	hipEvent_t startCuda, stopCuda;
	hipEventCreate(&startCuda);
	hipEventCreate(&stopCuda);

	if (input.empty())
	{
		cout << "Obrazok sa musi volat image.jpg a byt v rovnakom priecinku ako exe subor" << std::endl;
		return;
	}

	cv::Mat output(input.rows, input.cols, CV_8UC1);

	high_resolution_clock::time_point start = high_resolution_clock::now();
	ImageCPU(input.ptr(), output.ptr(), input.step, output.step, input.cols, input.rows, size);
	high_resolution_clock::time_point stop = high_resolution_clock::now();
	duration<double> duration = stop - start;
	CPUGPUTime("CPU time: " + to_string(duration.count()) + " s\n");

	start = high_resolution_clock::now();
	ImageGPU(input.ptr(), output.ptr(), input.step, output.step, input.cols, input.rows, output.rows, size);
	stop = high_resolution_clock::now();
	duration = stop - start;
	CPUGPUTime("GPU time: " + to_string(duration.count()) + " s\n");

	printTime();

	//cv::imshow("original", input);
	//cv::imshow("processed", output);

	cv::waitKey();

}

int main()
{
	hipEventCreate(&startCuda);
	hipEventCreate(&stopCuda);
	timeCudaMemcpyh2d = 0;
	timeCudaMemcpyd2h = 0;
	timeCudaMalloc = 0;
	timeKernel = 0;

	cout << "CUDA version:   v" << CUDART_VERSION << endl;

	int devCount;
	hipGetDeviceCount(&devCount);
	hipDeviceProp_t GPU;
	hipGetDeviceProperties(&GPU, 0);
	cout << GPU.name << ": " << GPU.major << "." << GPU.minor << endl;
	cout << "  Global memory:   " << GPU.totalGlobalMem / (1024 * 1024) << "mb" << endl;
	cout << "  Shared memory:   " << GPU.sharedMemPerBlock / 1024 << "kb" << endl;
	cout << "  Constant memory: " << GPU.totalConstMem / 1024 << "kb" << endl;

	cout << "  Warp size:         " << GPU.warpSize << endl;
	cout << "  Threads per block: " << GPU.maxThreadsPerBlock << endl;
	cout << "  Max block dimensions: [ " << GPU.maxThreadsDim[0] << ", " << GPU.maxThreadsDim[1] << ", " << GPU.maxThreadsDim[2] << " ]" << endl;
	//cout << "  Max grid dimensions:  [ " << GPU.maxGridSize[0] << ", " << GPU.maxGridSize[1] << ", " << GPU.maxGridSize[2] << " ]" << endl;
	cout << endl << endl;

	cout << "Test 1 -matrix multiplication\nTest 2 -memory copy\nTest 3 -float operations\nTest 4 -fibonaci\nTest 5 -image\n" << endl;

	kernelInit << <1, 1024 >> > ();
	hipDeviceSynchronize();

	Settings settings;
	readConfig(settings);

	char input;

	for (int i = 1; i < 6; i++)
	{
		printTestNumber("****************************************************\n\nTest "+ to_string(i) + "\n");
		for (int j = 1; j < settings.repeat+1; j++)
		{

			if (i == 1)
			{
				printTestNumber(to_string(j) +" : matrix size: "+to_string(settings.test1Base) + "\n");
				MatrixMultiplication(settings.test1Base);
				settings.test1Base += settings.test1Increment;
			}

			else if (i == 2)
			{
				printTestNumber(to_string(j) + " : array of int length : " + to_string(settings.test2Base) + "\n");
				memoryCopy(settings.test2Base);
				if (j >= 9) break;
				settings.test2Base *= settings.test2Increment;
			}

			else if (i == 3)
			{
				if (j > 6) break;
				IN = settings.test3Base;
				printTestNumber(to_string(j) + " number of operations: " + to_string(OUT * IN) + "\n");
				floatComputing();
				settings.test3Base *= settings.test3Increment;
			}

			else if (i == 5)
			{
				if (j > 4) break;
				printTestNumber(to_string(j) + " number of images " + to_string(settings.test5Base) + "\n");
				processImage(settings.test5Base);
				if (j >= 9) break;
				settings.test5Base *= settings.test5Increment;
			}

		}
	}

	ofstream resultsFile;
	resultsFile.open("results.txt");
	resultsFile << ss.rdbuf();
	resultsFile.close();
/*
	while (true)
	{
		cout << ">" << " ";
		scanf(" %c", &input);
		if (input == '9')
		{
			ofstream resultsFile;
			resultsFile.open("results.txt");
			resultsFile << ss.rdbuf();
			resultsFile.close();
			return 0;
		}

		if (input == '1')
		{
			cout << "\nmatrix size: ";

			int size;
			scanf(" %d", &size);
			MatrixMultiplication(size);
		}
		if (input == '2')
		{
			cout << "\narray size: ";

			int size;
			scanf(" %d", &size);
			memoryCopy(size);
		}
		if (input == '3')
		{
			floatComputing();
		}
		if (input == '4')
		{
			int size;
			scanf(" %d", &size);
			fibonaci(size);
		}

		if (input == '5')
		{
			processImage(100);
		}
	}*/

	cin.get();

	return 0;
}
